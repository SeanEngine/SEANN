#include "hip/hip_runtime.h"
//
// Created by DanielSun on 2/28/2022.
//

#include "TensorTools.cuh"
#include "GEMM.cuh"

#define toFloat4R(ptr) (reinterpret_cast<float4*>(&(ptr))[0])

namespace seblas{
    __global__ void extract4D(Tensor* in, Tensor* buffer, shape4 start, shape4 extractRange){
        unsigned int readRow = start.rows + threadIdx.y + blockIdx.y * blockDim.y;
        unsigned int readCol = start.cols + 4*(threadIdx.x + blockIdx.x * blockDim.x);
        shape4 src = in->dims;
        shape4 out = start + extractRange;
        #pragma unroll
        for(unsigned int n = start.n; n < out.n; n++){
            #pragma unroll
            for(unsigned int c = start.c; c < out.c; c++){
                if(readRow < extractRange.rows && readCol < extractRange.cols){
                    toFloat4R(buffer->elements[extractRange[
                            shape4(n-start.n,c-start.c,readRow -start.rows, readCol - start.cols)]])
                    = toFloat4R(in->elements[src[shape4(n,c,readRow,readCol)]]);
                }
            }
        }
    }

    Tensor* slice(Tensor *in, Tensor *buffer, DimPrefix dim, range dimRange) {
        switch (dim) {
            case DEPTH:
                assert(dimRange.b < in->dims.n);
                hipMemcpy(buffer->elements, in->elements + dimRange.a * in->dims.c * in->dims.rows * in->dims.cols
                        , sizeof(float) * (dimRange.b - dimRange.a) * in->dims.c * in->dims.rows * in->dims.cols,
                        hipMemcpyDeviceToDevice);
                break;
            case ROW:
                assert(dimRange.b < in->dims.c);
                hipMemcpy(buffer->elements, in->elements + dimRange.a * in->dims.rows * in->dims.cols
                        , sizeof(float) * (dimRange.b - dimRange.a) * in->dims.rows * in->dims.cols,
                           hipMemcpyDeviceToDevice);
                break;
            case COL:
                assert(dimRange.b < in->dims.rows);
                hipMemcpy(buffer->elements, in->elements + dimRange.a * in->dims.cols
                        , sizeof(float) * (dimRange.b - dimRange.a) * in->dims.cols,
                           hipMemcpyDeviceToDevice);
                break;
        }
        return buffer;
    }

    Tensor* extract4(Tensor *in, Tensor *buffer, shape4 start, shape4 extractRange) {
        assert(extractRange.cols%4==0);
        dim3 block = CUDA_BLOCK_SIZE;
        dim3 grid = ((extractRange.cols + block.x - 1)/(block.x * 4),
                (extractRange.rows + block.y - 1)/block.y);
        extract4D<<<grid, block>>>(in, buffer, start, extractRange);
        hipDeviceSynchronize();
        ErrorHandler::checkDeviceStatus(__FILE__, __LINE__);
        return buffer;
    }
}
