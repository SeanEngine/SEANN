#include "hip/hip_runtime.h"
//
// Created by DanielSun on 2/12/2022.
//

#include <cassert>
#include <iostream>
#include "GEMM.cuh"
#include "../assist/DBGTools.cuh"

using namespace seblas;

#define assertGemm(A,B,C) assert((A)->dims.cols == (B)->dims.rows \
&& (A)->dims.rows == (C)->dims.rows && (B)->dims.cols==(C)->dims.cols)

#define BM 128
#define BN 128
#define BK 8
#define RM 8
#define RN 8

#define toFloat4(ptr) (reinterpret_cast<float4*>(&(ptr))[0])

__global__ void gemmNaive(Tensor *A, Tensor *B, Tensor *C){
    unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int col = threadIdx.x + blockDim.x * blockIdx.x;
    float value = 0;
    if(row < C->dims.rows && col < C->dims.cols){
        for (int am = 0; am < A->dims.cols; am++) {
            value += A->getD(row, am) * B->getD(am,col);
        }
        C->setD(row,col,value);
    }
}

template<const int BLOCK_M, const int BLOCK_N, const int BLOCK_K,
        const int REGIS_M, const int REGIS_N>
__global__ void gemmPrefetchingSR(Tensor *A, Tensor *B, Tensor *C){
    const unsigned int M = A->dims.rows;
    const unsigned int N = B->dims.cols;
    const unsigned int K = A->dims.cols;

    ///allocate smems and registers
    //The shared memory tile
    __shared__ float tileA[2][BLOCK_K][BLOCK_M];  //transposed
    __shared__ float tileB[2][BLOCK_K][BLOCK_N];

    float regisA[2][REGIS_M];
    float regisB[2][REGIS_N];
    float regisC[REGIS_M][REGIS_N] = {0};

    const int threadDimX = BLOCK_N / REGIS_N;
    const int threadDimY = BLOCK_M / REGIS_M;
    const int threadCount = threadDimX * threadDimY;
    const int tid = threadIdx.y * threadDimX + threadIdx.x;

    ///register for buffering elements during transporting global to shared mem
    float bufferA[BLOCK_M * BLOCK_K / threadCount] = {0};
    float bufferB[BLOCK_N * BLOCK_K / threadCount] = {0};

    ///prepare configs for reading global
    float* ptrA = A->elements + blockIdx.y * BLOCK_M * K;
    float* ptrB = B->elements + blockIdx.x * BLOCK_N;
    const int blockM = blockIdx.y * BLOCK_M;
    const int blockN = blockIdx.x * BLOCK_N;

    const int readThreadPerRowA = BLOCK_K;
    const int readThreadPerRowB = BLOCK_N;

    //the location each thread should be reading relative to smem
    const int readRowA = tid / readThreadPerRowA;
    const int readColA = tid % readThreadPerRowA;

    const int readRowB = tid / readThreadPerRowB;
    const int readColB = tid % readThreadPerRowB;

    //these values are used to determine the amount of rows to jump
    //if there is the need to do read multiple times
    const int readRowStrideA = threadCount / readThreadPerRowA;
    const int readRowStrideB = threadCount / readThreadPerRowB;

    #pragma unroll
    for(int i=0; i<BLOCK_M; i+= readRowStrideA){
        if(blockM + readRowA + i < M && readColA < K){
            tileA[0][readColA][readRowA+i] = ptrA[(readRowA + i)*K + readColA];
        }
    }

    #pragma unroll
    for(int i=0; i<BLOCK_K; i+= readRowStrideB){
        if(readRowB + i< K && blockN + readColB < N){
            tileB[0][readRowB+i][readColB] = ptrB[(readRowB + i)*N + readColB];
        }
    }
    __syncthreads();

    #pragma unroll
    for(int rm = 0; rm < REGIS_M; rm += 4){
        toFloat4(regisA[0][rm]) = toFloat4(tileA[0][0][REGIS_M * threadIdx.y + rm]);
    }

    #pragma unroll
    for(int rn = 0; rn < REGIS_N; rn += 4){
        toFloat4(regisB[0][rn]) = toFloat4(tileB[0][0][REGIS_N * threadIdx.x + rn]);
    }

    ///main loop
    int writeStageFlag = 1;
    #pragma unroll
    for(int nextTileID = BLOCK_K; nextTileID < K + BLOCK_K - 1; nextTileID+=BLOCK_K) {
        //prefetch
        if (nextTileID < K) {
            #pragma unroll
            for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                int loadIndex = i / readRowStrideA;
                bufferA[loadIndex] = blockM + readRowA + i < M && readColA + nextTileID < K ?
                        ptrA[(readRowA + i) * K + readColA + nextTileID] : 0;
            }

            #pragma unroll
            for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                int loadIndex = i / readRowStrideB;
                bufferB[loadIndex] = readRowB + i + nextTileID < K && blockN + readColB < N ?
                        ptrB[(readRowB + i + nextTileID) * N + readColB] : 0;
            }
        }

        int nextStageFlag = writeStageFlag ^ 1;

        //compute the part that is already in the registers and load the next segment
        #pragma unroll
        for (int i = 0; i < BLOCK_K - 1; i++) {

            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm += 4) {
                toFloat4(regisA[(i + 1) % 2][rm]) = toFloat4(
                        tileA[nextStageFlag][i + 1][REGIS_M * threadIdx.y + rm]);
            }

            #pragma unroll
            for (int rn = 0; rn < REGIS_N; rn += 4) {
                toFloat4(regisB[(i + 1) % 2][rn]) = toFloat4(
                        tileB[nextStageFlag][i + 1][REGIS_N * threadIdx.x + rn]);
            }

            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm++) {
                #pragma unroll
                for (int rn = 0; rn < REGIS_N; rn++) {
                    regisC[rm][rn] += regisA[i % 2][rm] * regisB[i % 2][rn];
                }
            }
        }

        //load the data in the register buffers to tiles
        if (nextTileID < K) {
            #pragma unroll
            for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                int loadIndex = i / readRowStrideA;
                tileA[writeStageFlag][readColA][readRowA + i] = bufferA[loadIndex];
            }

            #pragma unroll
            for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                int loadIndex = i / readRowStrideB;
                tileB[writeStageFlag][readRowB + i][readColB] = bufferB[loadIndex];
            }

            __syncthreads();
            writeStageFlag ^= 1;  //switch
        }
        #pragma unroll
        for (int rm = 0; rm < REGIS_M; rm += 4) {
            toFloat4(regisA[0][rm]) = toFloat4(
                    tileA[nextStageFlag ^ 1][0][REGIS_M * threadIdx.y + rm]);
        }

        #pragma unroll
        for (int rn = 0; rn < REGIS_N; rn += 4) {
            toFloat4(regisB[0][rn]) = toFloat4(
                    tileB[nextStageFlag ^ 1][0][REGIS_N * threadIdx.x + rn]);
        }

        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm ++){
            #pragma unroll
            for(int rn = 0; rn < REGIS_N; rn ++){
                regisC[rm][rn] += regisA[1][rm] * regisB[1][rn];
            }
        }
    }
    #pragma unroll
    for(int rm = 0; rm < REGIS_M; rm ++){
        #pragma unroll
        for(int rn = 0; rn < REGIS_N; rn ++){
            if((blockM + threadIdx.y * REGIS_M + rm < M && blockN + threadIdx.x * REGIS_N + rn < N)) {
                C->elements[(blockM + threadIdx.y * REGIS_M + rm) * N
                                     + blockN + threadIdx.x * REGIS_N + rn] = regisC[rm][rn];
            }
        }
    }
}

/**
 * The fast gemm that utilized smem and registers with data prefetching
 * @tparam BLOCK_M block size m
 * @tparam BLOCK_N block size n
 * @tparam BLOCK_K block size k
 * @tparam REGIS_M (the size of the sub matrix of C each thread compute : rows)
 * @tparam REGIS_N (the size of the sub matrix of C each thread compute : cols)
 * @param A
 * @param B
 * @param C
 */
template<const int BLOCK_M, const int BLOCK_N, const int BLOCK_K,
        const int REGIS_M, const int REGIS_N>
__global__ void gemmPrefetching(Tensor *A, Tensor *B, Tensor *C) {

    const unsigned int M = A->dims.rows;
    const unsigned int N = B->dims.cols;
    const unsigned int K = A->dims.cols;

    ///allocate smems and registers
    //The shared memory tile
    __shared__ float tileA[2][BLOCK_K][BLOCK_M];  //transposed
    __shared__ float tileB[2][BLOCK_K][BLOCK_N];

    float regisA[2][REGIS_M];
    float regisB[2][REGIS_N];
    float regisC[REGIS_M][REGIS_N] = {0};

    const int threadDimX = BLOCK_N / REGIS_N;
    const int threadDimY = BLOCK_M / REGIS_M;
    const int threadCount = threadDimX * threadDimY;
    const int tid = threadIdx.y * threadDimX + threadIdx.x;

    ///register for buffering elements during transporting global to shared mem
    float bufferA[BLOCK_M * BLOCK_K / threadCount] = {0};
    float bufferB[BLOCK_N * BLOCK_K / threadCount] = {0};

    ///prepare configs for reading global
    float* ptrA = A->elements + blockIdx.y * BLOCK_M * K;
    float* ptrB = B->elements + blockIdx.x * BLOCK_N;
    const int blockM = blockIdx.y * BLOCK_M;
    const int blockN = blockIdx.x * BLOCK_N;

    const int readThreadPerRowA = BLOCK_K / 4;
    const int readThreadPerRowB = BLOCK_N / 4;

    //the location each thread should be reading relative to smem
    const int readRowA = tid / readThreadPerRowA;
    const int readColA = tid % readThreadPerRowA * 4;

    const int readRowB = tid / readThreadPerRowB;
    const int readColB = tid % readThreadPerRowB * 4;

    //these values are used to determine the amount of rows to jump
    //if there is the need to do read multiple times
    const int readRowStrideA = threadCount / readThreadPerRowA;
    const int readRowStrideB = threadCount / readThreadPerRowB;


    ///prefetch the first smem and register block before starting the main loop
    #pragma unroll
    for(int i = 0; i < BLOCK_M; i+=readRowStrideA){
        int loadIndex = i / readRowStrideA * 4;
        if(blockM + readRowA + i < M && readColA < K) {
            toFloat4(bufferA[loadIndex]) = toFloat4(ptrA[(readRowA + i) * K + readColA]);
            //transpose
            tileA[0][readColA][readRowA + i] = bufferA[loadIndex];
            tileA[0][readColA + 1][readRowA + i] = bufferA[loadIndex + 1];
            tileA[0][readColA + 2][readRowA + i] = bufferA[loadIndex + 2];
            tileA[0][readColA + 3][readRowA + i] = bufferA[loadIndex + 3];
        }
    }

    #pragma unroll
    for(int i = 0; i < BLOCK_K; i+=readRowStrideB){
        if(readRowB + i < K && blockN + readColB < N){
            toFloat4(tileB[0][readRowB + i][readColB]) = toFloat4(ptrB[(readRowB + i)*N + readColB]);
        }
    }
    __syncthreads();

    #pragma unroll
    for(int rm = 0; rm < REGIS_M; rm += 4){
        toFloat4(regisA[0][rm]) = toFloat4(tileA[0][0][REGIS_M * threadIdx.y + rm]);
    }

    #pragma unroll
    for(int rn = 0; rn < REGIS_N; rn += 4){
        toFloat4(regisB[0][rn]) = toFloat4(tileB[0][0][REGIS_N * threadIdx.x + rn]);
    }

    ///main loop
    int writeStageFlag = 1;
    #pragma unroll
    for(int nextTileID = BLOCK_K; nextTileID < K + BLOCK_K - 1; nextTileID+=BLOCK_K){
        //prefetch
        if(nextTileID < K) {
            #pragma unroll
            for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                int loadIndex = i / readRowStrideA * 4;
                if (blockM + readRowA + i < M && readColA + nextTileID < K) {
                    toFloat4(bufferA[loadIndex]) = toFloat4(
                            ptrA[(readRowA + i) * K + readColA + nextTileID]);
                }else{
                    bufferA[loadIndex] = 0;
                    bufferA[loadIndex+1] = 0;
                    bufferA[loadIndex+2] = 0;
                    bufferA[loadIndex+3] = 0;
                }
            }

            #pragma unroll
            for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                int loadIndex = i / readRowStrideB * 4;
                if (readRowB + i + nextTileID < K && blockN + readColB < N) {
                    toFloat4(bufferB[loadIndex]) = toFloat4(
                            ptrB[(readRowB + i + nextTileID) * N + readColB]);
                } else {
                    bufferA[loadIndex] = 0;
                    bufferA[loadIndex+1] = 0;
                    bufferA[loadIndex+2] = 0;
                    bufferA[loadIndex+3] = 0;
                }
            }
        }

        int nextStageFlag = writeStageFlag ^ 1;

        //compute the part that is already in the registers and load the next segment
        #pragma unroll
        for(int i = 0; i < BLOCK_K-1; i++){

            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm += 4) {
                toFloat4(regisA[(i + 1) % 2][rm]) = toFloat4(
                        tileA[nextStageFlag][i + 1][REGIS_M * threadIdx.y + rm]);
            }

            #pragma unroll
            for (int rn = 0; rn < REGIS_N; rn += 4) {
                toFloat4(regisB[(i + 1) % 2][rn]) = toFloat4(
                        tileB[nextStageFlag][i + 1][REGIS_N * threadIdx.x + rn]);
            }

            #pragma unroll
            for(int rm = 0; rm < REGIS_M; rm ++){
                #pragma unroll
                for(int rn = 0; rn < REGIS_N; rn ++){
                    regisC[rm][rn] += regisA[i%2][rm] * regisB[i%2][rn];
                }
            }
        }

        //load the data in the register buffers to tiles
        if(nextTileID < K){
            #pragma unroll
            for(int i=0; i<BLOCK_M; i+=readRowStrideA){
                int loadIndex = i/readRowStrideA * 4;
                tileA[writeStageFlag][readColA][readRowA + i] = bufferA[loadIndex];
                tileA[writeStageFlag][readColA + 1][readRowA + i] = bufferA[loadIndex + 1];
                tileA[writeStageFlag][readColA + 2][readRowA + i] = bufferA[loadIndex + 2];
                tileA[writeStageFlag][readColA + 3][readRowA + i] = bufferA[loadIndex + 3];
            }

            #pragma unroll
            for(int i = 0; i < BLOCK_K; i+=readRowStrideB){
                int loadIndex = i/readRowStrideA * 4;
                toFloat4(tileB[writeStageFlag][readRowB + i][readColB]) = toFloat4(bufferB[loadIndex]);
            }

            __syncthreads();
            writeStageFlag ^= 1;  //switch
        }

        #pragma unroll
        for (int rm = 0; rm < REGIS_M; rm += 4) {
            toFloat4(regisA[0][rm]) = toFloat4(
                    tileA[nextStageFlag ^ 1][0][REGIS_M * threadIdx.y + rm]);
        }

        #pragma unroll
        for (int rn = 0; rn < REGIS_N; rn += 4) {
            toFloat4(regisB[0][rn]) = toFloat4(
                    tileB[nextStageFlag ^ 1][0][REGIS_N * threadIdx.x + rn]);
        }

        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm ++){
            #pragma unroll
            for(int rn = 0; rn < REGIS_N; rn ++){
                regisC[rm][rn] += regisA[1][rm] * regisB[1][rn];
            }
        }
    }

    #pragma unroll
    for(int rm = 0; rm < REGIS_M; rm ++){
        #pragma unroll
        for(int rn = 0; rn < REGIS_N; rn += 4){
            if((blockM + threadIdx.y * REGIS_M + rm < M && blockN + threadIdx.x * REGIS_N + rn < N)) {
                toFloat4(C->elements[(blockM + threadIdx.y * REGIS_M + rm) * N
                                     + blockN + threadIdx.x * REGIS_N + rn]) = toFloat4(regisC[rm][rn]);
            }
        }
    }
}

Tensor *seblas::callGemmPrefetching(Tensor *A, Tensor *B, Tensor *C) {

    assert(A->dims.cols == B->dims.rows);
    assert(A->dims.rows == C->dims.rows && B->dims.cols == C->dims.cols);


    dim3 grid = dim3((C->dims.cols + BN - 1) / BN, (C->dims.rows + BM - 1) / BM);
    dim3 block = dim3(BN / RN, BM / RM);

    if(A->dims.cols%4==0 && B->dims.cols%4==0){
        gemmPrefetching<BM, BN, BK, RM, RN><<<grid, block>>>(A,B,C);
    } else {
        gemmPrefetchingSR<BM, BN, BK, RM, RN><<<grid, block>>>(A,B,C);
    }
    hipDeviceSynchronize();
    ErrorHandler::checkDeviceStatus(__FILE__,__LINE__);
    return C;
}

Tensor* seblas::callGemmNaive(Tensor* A, Tensor* B, Tensor* C){
    assertGemm(A,B,C);
    dim3 grid = dim3((C->dims.cols + CUDA_BLOCK_SIZE.x-1)/CUDA_BLOCK_SIZE.x ,
                     (C->dims.rows + CUDA_BLOCK_SIZE.y-1)/CUDA_BLOCK_SIZE.y);
    gemmNaive<<<grid, CUDA_BLOCK_SIZE>>>(A,B,C);
    hipDeviceSynchronize();
    ErrorHandler::checkDeviceStatus(__FILE__,__LINE__);
    return C;
}