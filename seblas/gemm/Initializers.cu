#include "hip/hip_runtime.h"
//
// Created by DanielSun on 3/22/2022.
//

#include <hiprand/hiprand_kernel.h>
#include "Initializers.cuh"
namespace seblas{
    //this method will generate a pair of random floats subjecting to normal distribution
    __global__ void randNormalD(Tensor* A, float mean, float stddev, long seed){
        uint32 id = (threadIdx.x + blockIdx.x * blockDim.x);
        if(id >= A->dims.size) return;
        hiprandStateXORWOW_t state;
        hiprand_init(id * seed, 0, 0, &state);
        float val = hiprand_normal(&state);
        A->elements[id] = val * stddev + mean;
    }

    __global__ void randUniformD(Tensor* A, float min, float max, long seed){

        uint32 id = (threadIdx.x + blockIdx.x * blockDim.x);
        if(id >= A->dims.size) return;
        hiprandStateXORWOW_t state;
        hiprand_init(id * seed, 0, 0, &state);
        float val = hiprand_uniform(&state);
        A->elements[id] = val * (max - min) + min;
    }

    Tensor* randNormal(Tensor* A, float mean, float stddev){
        long seed = (long)duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
        uint32 block = CUDA_BLOCK_SIZE.y * CUDA_BLOCK_SIZE.x;
        uint32 grid = (A->dims.size + block - 1) / block;
        randNormalD<<<grid, block>>>(A, mean, stddev, seed);
        hipDeviceSynchronize();
        ErrorHandler::checkDeviceStatus(__FILE__, __LINE__);
        return A;
    }

    Tensor* randUniform(Tensor* A, float min, float max){
        long seed = (long)duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
        uint32 block = CUDA_BLOCK_SIZE.y * CUDA_BLOCK_SIZE.x;
        uint32 grid = (A->dims.size + block - 1) / block;
        randUniformD<<<grid, block>>>(A, min, max, seed);
        hipDeviceSynchronize();
        ErrorHandler::checkDeviceStatus(__FILE__, __LINE__);
        return A;
    }
}