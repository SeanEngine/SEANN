//
// Created by DanielSun on 2/12/2022.
//

#include "ErrorHandler.cuh"
#include <iostream>
#include <cassert>


void seblas::ErrorHandler::checkDeviceStatus(const char* file, int line) {
     hipError_t err = hipGetLastError();
     if(err!=hipSuccess){
         seio::logFatal(seio::LOG_SEG_SEBLAS,string("Encountered CUDA Errors : ") + " line: " +
         to_string(line) + " " + hipGetErrorString(err) + "\n" + file);
         assert(err==hipSuccess);
     }
}
