//
// Created by DanielSun on 2/12/2022.
//

#include "ErrorHandler.cuh"
#include <iostream>
#include <cassert>

void seblas::ErrorHandler::checkDeviceStatus() {
     hipError_t err = hipGetLastError();
     if(err!=hipSuccess){
         std::cout<<"Encountered CUDA Errors : "<<__FILE__<<" "<<__LINE__<<hipGetErrorString(err)<<std::endl;
         assert(err==hipSuccess);
     }
}
