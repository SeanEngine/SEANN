//
// Created by DanielSun on 2/12/2022.
//

#include "ErrorHandler.cuh"
#include <iostream>
#include <cassert>

void seblas::ErrorHandler::checkDeviceStatus(const char* file, int line) {
     hipError_t err = hipGetLastError();
     if(err!=hipSuccess){
         std::cout<<"Encountered CUDA Errors : "<<file<<" "<<line<<" "<<hipGetErrorString(err)<<std::endl;
         assert(err==hipSuccess);
     }
}
