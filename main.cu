#include <iostream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include "seblas/gemm/GEMM.cuh"
#include "hip/hip_runtime.h"
#include "seblas/gemm/Tensor.cuh"
#include "seblas/gemm/TensorTools.cuh"

#include "hipblas.h"
#include "seblas/assist/DBGTools.cuh"
#include "seio/loader/ImageReader.cuh"
#pragma comment(lib, "cublas.lib")


using namespace seblas;
using namespace seio;
using namespace std;

int main(int argc, char **argv) {
    auto* A = Tensor::declare(24,12)->create()->randomFill();
    auto* B = Tensor::declare(12,24)->create();

    transpose(A,B);
    inspect(B);
}