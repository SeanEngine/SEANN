#include <iostream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include "seblas/gemm/GEMM.cuh"
#include "hip/hip_runtime.h"
#include "seblas/gemm/Tensor.cuh"
#include "seblas/gemm/TensorTools.cuh"

#include "hipblas.h"
#include "seblas/assist/DBGTools.cuh"
#include "seio/loader/ImageReader.cuh"
#pragma comment(lib, "cublas.lib")


using namespace seblas;
using namespace seio;
using namespace std;

int main(int argc, char **argv) {
     auto* A = Tensor::declare(3,1,3,3)->create()->randomFill();
     auto* B = Tensor::declare(3,32,32)->create()->randomFill();
     auto* C = Tensor::declare(1,32,32)->create();

     convDeriv(A,B,C,1,1,1);
     inspect(C);
}