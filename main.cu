#include <iostream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include "seblas/gemm/GEMM.cuh"
#include "hip/hip_runtime.h"
#include "seblas/gemm/Tensor.cuh"
#include "seblas/gemm/TensorTools.cuh"
#include "seblas/gemm/NeuralUtils.cuh"

#include "hipblas.h"
#include "seblas/assist/DBGTools.cuh"
#include "seio/loader/ImageReader.cuh"
#pragma comment(lib, "cublas.lib")


using namespace seblas;
using namespace seio;
using namespace std;

int main(int argc, char **argv) {
    auto* A = Tensor::declare(3, 32,32)->create()->constFill(3);
    auto* B = Tensor::declare(3, 1)->create()->constFill(-1);
    auto* C = Tensor::declare(3, 5)->create();

    convBiasError(A,B,C);
    inspect(B);
}