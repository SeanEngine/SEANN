#include <iostream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include "seblas/gemm/GEMM.cuh"
#include "hip/hip_runtime.h"
#include "seblas/gemm/Tensor.cuh"
#include "seblas/gemm/TensorTools.cuh"
#include "seblas/gemm/NeuralUtils.cuh"

#include "hipblas.h"
#include "seblas/assist/DBGTools.cuh"
#include "seio/loader/ImageReader.cuh"
#pragma comment(lib, "cublas.lib")


using namespace seblas;
using namespace seio;
using namespace std;

int main(int argc, char **argv) {
     auto* A = Tensor::declare(1,1025)->create()->constFill(4);
     auto* B = Tensor::declare(1,1025)->create();
    inspect(softmax(A, B));
}