#include <iostream>
#include <windows.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "seblas/gemm/GEMM.cuh"
#include "hip/hip_runtime.h"
#include "seblas/gemm/Tensor.cuh"
#include "seblas/gemm/TensorTools.cuh"
#include "seblas/gemm/NeuralUtils.cuh"

#include "hipblas.h"
#include "seblas/assist/DBGTools.cuh"
#include "seio/loader/DataLoader.cuh"
#include "seio/logging/LogUtils.cuh"
#include "sexec/threading/ThreadController.cuh"
#include "seann/models/testModel/ConvSTD.cuh"
#pragma comment(lib, "cublas.lib")

using namespace sexec;
using namespace seblas;
using namespace seann;
using namespace seio;
using namespace std;

int main(int argc, char **argv) {

    auto* model = new ConvSTD();
    model->registerModel();
    model->loadDataset();
}