#include <iostream>
#include <windows.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "seblas/gemm/GEMM.cuh"
#include "hip/hip_runtime.h"
#include "seblas/gemm/Tensor.cuh"
#include "seblas/gemm/TensorTools.cuh"
#include "seblas/gemm/NeuralUtils.cuh"

#include "hipblas.h"
#include "seblas/assist/DBGTools.cuh"
#include "seio/loader/ImageReader.cuh"
#include "seio/logging/LogUtils.cuh"
#pragma comment(lib, "cublas.lib")


using namespace seblas;
using namespace seio;
using namespace std;

int main(int argc, char **argv) {
    logInfo(seio::LOG_SEG_SEANN, "Test");
    return 0;
}